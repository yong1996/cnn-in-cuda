#include "hip/hip_runtime.h"
#include "layer.h"

#define MASK_WIDTH = 3;

// Layer constructor:
Layer::Layer(int in_width, int in_height, int in_size): width(in_width), height(in_height), bytes(in_size){

    float h_bias[in_height];
    float h_weight[height][width];

    output = NULL;
    preact = NULL;
    bias = NULL:
    weight = NULL;

    for (int i = 0; i < height; i++){
        h_bias[i] = 0.5f - float(rand()) / float(RAND_MAX);  // initial bias
        for (int j = 0; j < width; j++){
            h_weight[i][j] = 0.5f - float(rand()) / float(RAND_MAX);  // initial weight
        }
    }

    hipMalloc(&output, sizeof(float) * bytes);
	hipMalloc(&preact, sizeof(float) * bytes);

	hipMalloc(&bias, sizeof(float) * N);
	hipMalloc(&weight, sizeof(float) * M * N);

	hipMalloc(&d_output, sizeof(float) * bytes);
	hipMalloc(&d_preact, sizeof(float) * bytes);
	hipMalloc(&d_weight, sizeof(float) * M * N);

	hipMemcpy(bias, h_bias, sizeof(float) * N, hipMemcpyHostToDevice);
	hipMemcpy(weight, h_weight, sizeof(float) * M * N, hipMemcpyHostToDevice);
}

// de-constructor
Layer::~Layer(){

    // TODO: free cuda memory
    hipFree(output);
	hipFree(preact);

	hipFree(bias);

	hipFree(weight);

	hipFree(d_output);
	hipFree(d_preact);
	hipFree(d_weight);

}

void Layer:: output(float *data){
    hipMemcpy(output, data, sizeof(float)*bytes, hipMemcpyHostToDevice);
}

void Layer:: clear(){

}
void Layer:: bp_clear(){

}


__device__ float sigmoid(float v){
    return 1/(1 + exp(-v));
}

__global__ void step_function(float *input, float *output, const int N){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int size = blockDim.x * gridDim.x;
    // TODO:

}

// convLayer 1 the weight is 6*3*3  output is 6*27*27
__global__ void convLayer_forward(int C, int W_grid, int K, float *input[28][28], float output[6][27][27], float weight[6][3][3]){

    
    // int m, h, w, c, q, p;
    // float Pvalue = 0;
    // int n_start_point = i - (MASK_WIDTH/2);

    // for (int m = 0; m < 6; m++){          // for each output feature map
    //     for (int h = 0; h < 27; h++ ){
    //         for(int w = 0; w < 27; w++){
    //             output[m][h][w] = 0;
    //             for (int c = 0; c < 1; c++){ // for each channel
    //                 for (int p = 0; p < MASK_WIDTH; p++){
    //                     for (int q = 0; q < MASK_WIDTH; q++){
    //                         output[m][h][w] += input[h + p][w + q] * weight[m][p][q];
    //                     }
    //                 }
    //             }
    //         }

    //     }

    // }

    int m = blockIdx.x;
    int h =  blockIdx.y / W_grid  + threadIdx.y;
    int w = blockIdx.y % W_grid + threadIdx.x;
    float acc = 0.;
    //for (int c = 0;  c < C; c++) {		// sum over all input channels, in this case, the channel is 1
       for (int p = 0; p < K; p++)		// loop over KxK  filter
          for (int q = 0; q < K; q++)  
             acc += X[h+p][w+q] * W[m][p][q];
    //}
    output[m][h][w] = acc;

}


