#include "hip/hip_runtime.h"
#include "layer.h"


#define TILE_WIDTH 16


// Layer constructor:
Layer::Layer(int in_width, int in_height, int in_size): M(in_width), N(in_height), bytes(in_size){

    float h_bias[N];
    float h_weight[N][M];


    output = NULL;
    preact = NULL;
    bias = NULL;
    weight = NULL;

    for (int i = 0; i < N; i++){
        h_bias[i] = 0.5f - float(rand()) / float(RAND_MAX);  // initial bias
        for (int j = 0; j < M; j++){
            h_weight[i][j] = 0.5f - float(rand()) / float(RAND_MAX);  // initial weight
        }
    }

    hipMalloc(&output, sizeof(float) * bytes);
	hipMalloc(&preact, sizeof(float) * bytes);

	hipMalloc(&bias, sizeof(float) * N);
	hipMalloc(&weight, sizeof(float) * M * N);

	hipMalloc(&d_output, sizeof(float) * bytes);
	hipMalloc(&d_preact, sizeof(float) * bytes);
	hipMalloc(&d_weight, sizeof(float) * M * N);

	hipMemcpy(bias, h_bias, sizeof(float) * N, hipMemcpyHostToDevice);
	hipMemcpy(weight, h_weight, sizeof(float) * M * N, hipMemcpyHostToDevice);
}

// de-constructor
Layer::~Layer(){

    // TODO: free cuda memory
    hipFree(output);
	hipFree(preact);

	hipFree(bias);

	hipFree(weight);

	hipFree(d_output);
	hipFree(d_preact);
	hipFree(d_weight);

}

void Layer:: setInput(float *data){
    hipMemcpy(output, data, sizeof(float)*bytes, hipMemcpyHostToDevice);
}

// Reset GPU memory between iterations
void Layer::clear()
{
	hipMemset(output, 0x00, sizeof(float) * bytes);
	hipMemset(preact, 0x00, sizeof(float) * bytes);
}


void Layer::bp_clear()
{
	hipMemset(d_output, 0x00, sizeof(float) * bytes);
	hipMemset(d_preact, 0x00, sizeof(float) * bytes);
	hipMemset(d_weight, 0x00, sizeof(float) * M * N);
}


__device__ float sigmoid(float s){
    return 1/(1 + exp(-s));
}

__global__ void apply_sigmoid(float *input, float *output, const int N){
    int pos = blockIdx.x * blockDim.x + threadIdx.x;
    int size = blockDim.x * gridDim.x;
    // TODO:
    for (int idx = N * pos / size; idx < N * (pos+1) / size; ++idx) {
		output[idx] = sigmoid(input[idx]);
	}
}

// __global__ void backward_sigmoid(float* X, int size_in)
// {
// 	int t = blockIdx.x * 1024 + threadIdx.x;

// 	if(t < size_in)
// 	{
// 		double tmp = 1 / (1 + exp(-X[t]));
// 		tmp = (1-tmp)*tmp;
// 		X[t] = X[t]*tmp;
// 	}
// }


__global__ void makeError(float *err, float *output, unsigned int Y, const int N)
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	for (int idx = N * pos / size; idx < N * (pos+1) / size; ++idx) {
		err[idx] = ((Y == idx ? 1.0f : 0.0f) - output[idx]);
	}
}

#define TILE_WIDTH 16

//input_pointer,  Output_pointer, W_pointer, Inputimage_channel, Inputimage_height, Inputimage_width , Outputimage_width, W_width_height, Outputimage_channel
__global__ void ConvLayerForward_Kernel_1(float input[28][28], float output[6][24][24], float weight[6][5][5], float bias[6], int C, int H_in, int W_in, int W_out, int K, int M){
    int H_out = H_in - K + 1;
	int W_grid = ceilf((float)W_out/TILE_WIDTH);
	if(W_grid==0)
		W_grid = 1;
	// int n = blockIdx.x;
	int m = blockIdx.y;
	int h = (blockIdx.z / W_grid)*TILE_WIDTH + threadIdx.y;
	int w = (blockIdx.z % W_grid)*TILE_WIDTH + threadIdx.x;

	float acc = 0;
	int c, p, q;
	for (c = 0; c < C; c++) { // sum over all input channels
		for (p = 0; p < K; p++) // loop over KxK filter
			for (q = 0; q < K; q++)
				if(h < H_out && w < W_out)
                    acc += input[h+p][w+q] * weight[m][p][q];
					//acc = acc + X[n*(C*H_in*W_in) + c*(H_in*W_in) + (h+p)*(W_in) + (w+q)] * W[m*(C*K*K) + c*(K*K) + p*(K) + q];
	}
	__syncthreads();
	if(h < H_out && w < W_out)
	{
        output[m][h][w] = acc + bias[m];
    }
}


// input_pointer, output_pointer, inputimage_height, inputimage_width, outputimage_channel, pool_size 
__global__ void MaxPool2dForward_Kernel_1(float input[6][24][24], float output[6][6][6], float weight[1][4][4], float bias[1] ,int H_in, int W_in, int M, int pool_size){
	int H_out = H_in/pool_size;
	int W_out = W_in/pool_size;
	int W_grid = ceilf((float)W_out/TILE_WIDTH);
	if(W_grid==0){
        W_grid = 1;
    }
		
	// int n = blockIdx.x;
	int m = blockIdx.y;
	int h = (blockIdx.z / W_grid)*TILE_WIDTH + threadIdx.y;
	int w = (blockIdx.z % W_grid)*TILE_WIDTH + threadIdx.x;
	//h and w is not center point of calculating, it's upper left corner point of Input image
	
	float acc = 0;
	int p, q;
	for (p = 0; p < pool_size; p++) { // loop over KxK input samples
		for (q = 0; q < pool_size; q++)
			if(h < H_out && w < W_out)
				// acc = acc + input[n*(M*H_in*W_in)+ m*(H_in*W_in) +
				//               (pool_size * h + p)*(W_in) + (pool_size * w + q)] / (pool_size * pool_size);
                acc = acc + input[m][pool_size * h+p][pool_size * w+q] * weight[0][p][q];
	}
	__syncthreads();
	if(h < H_out && w < W_out)
	{
		// Y[n*(M*H_out*W_out)+ m*(H_out*W_out) + h*(W_out) + w] = acc;
		output[m][h][w] = acc + bias[0];

	}
}




// input_height, input_width, weight_width, output_height, output_width
//      1             6          10          1              10
__global__ void FullyConLayerForward_kernel(float input[6][6][6], float weight[10][6][6][6], float output[10], float bias[10], int H_in, int W_in, int W_we , int H_out, int W_out) {
	int W_grid = ceilf((float)W_out/TILE_WIDTH);
	if(W_grid==0)
		W_grid = 1;

	// int n = blockIdx.x;
	int m = blockIdx.y;  // 10
	int h = threadIdx.x;  // 6
	int w = threadIdx.y;  // 6
	int y = threadIdx.z;  // 6

	float Pvalue = 0;
	int o, p, q;
	for (o = 0; o < 6; o++) {
		for (p = 0; p < 6; p++) {
			for (q = 0; q < 6; q++){
				if(h < 6 && w < 6 && y < 6)
				// Pvalue += input[y][h+p][w+q] * weight[m][y][h+p][w+q];
				// Pvalue += input[h][w][y] * weight[m][h+o][w+p][y+q];
				Pvalue+= input[o][p][q] * weight[m][o][p][q];
			}
		}
	}
	__syncthreads();

    if(m < W_out && h < 6 && w < 6 && y < 6)
		output[m] = Pvalue + bias[m]; // Output


}




// __global__ void FullyConLayerBackward_kernel(
// 	float lf_output[10],
// 	float l_f_d_preact[10],
// 	float ls1_preact[6][6][6],
// 	float lf_weight[10][6][6][6],
// 	float lf_d_weight[10][6][6][6],
// 	float lf_bias[10]
// ) {
// 	// int n = blockIdx.x;
// 	int m = blockIdx.y;  // 10
// 	int h = threadIdx.x;  // 6
// 	int w = threadIdx.y;  // 6
// 	int y = threadIdx.z;  // 6

// 	l_f_d_preact[m] *= lf_output[m] * (1- lf_output[m]);
// 	__syncthreads();
// 	// ls1_d_preact[m] = l_f_d_preact[m] * lf_output[m] * (1- lf_output[m]);

// 	lf_bias[m] += dt + l_f_d_preact[m];
	
// 	lf_d_weight[m][h][w][y] = l_f_d_preact[m] * ls1_preact[h][w][y] ;
// 	lf_d_weight[m][h][w][y] += lf_weight[m][h][w][y];
// }


// //input_pointer, Inputimage_height, Inputimage_width, output_pointer, Outputimage_channel, pool_size
// __global__ void poolingLayer_backward_GPU(float input[6][24][24], int H_in, int W_in, float output[6][6][6], int M, int pool_size)

// {
// 	int H_out = H_in/pool_size;
// 	int W_out = W_in/pool_size;
// 	int W_grid = ceilf((float)W_out/TILE_WIDTH);
// 	if(W_grid==0)
// 		W_grid = 1;
// 	// int n = blockIdx.x;
// 	int m = blockIdx.y;
// 	int h = (blockIdx.z / W_grid)*TILE_WIDTH + threadIdx.y;
// 	int w = (blockIdx.z % W_grid)*TILE_WIDTH + threadIdx.x;

// 	//h and w is not center point of calculating, it's upper left corner point of Input image
// 	float acc = 0;
// 	for (int p = 0; p < pool_size; p++) { // loop over KxK input samples
// 		for (int q = 0; q < pool_size; q++)
// 			if(h < H_out && w < W_out)
// 			input[m][h+p][w+q] = output[m][h][w] / (pool_size * pool_size);
// 	}
// 	__syncthreads();

// }



// __global__ void ConvLayerBackward_Kernel(
// 	float input[28][28], 
// 	float d_output[6][24][24], 
// 	float preact[6][24][24], 
// 	float d_preact[6][24][24], 
// 	float d_weight[6][5][5], 
// 	int C, int H_in, int W_in, int W_out, int K, int M) {

//     int H_out = H_in - K + 1;
// 	int n, m, h, w, c, p, q;
// 	int W_grid = ceilf((float)W_out/TILE_WIDTH);
// 	if(W_grid==0)
// 		W_grid = 1;
// 	n = blockIdx.x;
// 	m = blockIdx.y;
// 	h = (blockIdx.z / W_grid)*TILE_WIDTH + threadIdx.y;
// 	w = (blockIdx.z % W_grid)*TILE_WIDTH + threadIdx.x;

// 	float d = 24.0f * 24.0f;

// 	float o = sigmoid(preact[m][h][w]);
	
// 	// float dv = d_output[m][h][w] * o * (1 - o);
// 	d_preact[m][h][w] = d_output[m][h][w] * o * (1 - o);
// 	__syncthreads();

// 	for (c = 0; c < C; c++) {
// 		for (p = 0; p < K; p++) {
// 			for (q = 0; q < K; q++) {
// 				if(h < H_out && w < W_out) {
// 					d_weight[m][p][q] = d_preact[m][h][w] * input[28][28]/d;
// 				}
// 			}
// 		}
// 	}
// }




// bp
// /*
// =========================

__global__ void bp_f(
	float l_f_d_weight[10][6][6][6],
	float l_f_d_preact[10],
	float l_f_bias[10],
	float l_f_weight[10][6][6][6],
	float l_s1_output[6][6][6],
	float l_s1_d_output[6][6][6],
	float l_s1_d_preact[6][6][6]
){
	// int n = blockIdx.x;
	int m = blockIdx.y;  // 10
	int h = threadIdx.x;  // 6
	int w = threadIdx.y;  // 6
	int y = threadIdx.z;  // 6


	l_f_d_weight[m][h][w][y] = l_f_d_preact[m] * l_s1_output[h][w][y];
	// l_s1_d_output[h][w][y] += l_f_weight[m][h][w][y] * l_f_d_preact[m];

	atomicAdd(&l_s1_d_output[h][w][y], l_f_weight[m][h][w][y] * l_f_d_preact[m]);
	if(h==0 && w==0 && y==0 )
		l_f_bias[m] += dt * l_f_d_preact[m];

	l_f_weight[m][h][w][y] += dt * l_f_d_weight[m][h][w][y];
}

__global__ void bp_s1(
	float l_s1_preact[6][6][6],
	float l_s1_d_output[6][6][6],
	float l_s1_d_preact[6][6][6],
	float l_s1_d_weight[1][4][4],
	float l_s1_weight[1][4][4],
	float l_c1_output[6][24][24],
	float l_c1_d_output[6][24][24],
	float l_s1_bias[6]
){
	// int n = blockIdx.x;
	int m = blockIdx.y;  // 6
	int h = threadIdx.x;  // 6
	int w = threadIdx.y;  // 6
	// int y = threadIdx.z;

	float o = sigmoid(l_s1_preact[m][h][w]);
	l_s1_d_preact[m][h][w] = l_s1_d_output[m][h][w] * o * (1 - o);

	// l_s1_d_preact[m][h][w] = l_s1_d_output[m][h][w] * l_s1_output[m][h][w] * (1 - l_s1_output[m][h][w]);
	__syncthreads();

	l_s1_bias[0] += dt * l_s1_d_preact[m][h][w]/(6*6*6);

	int i,j;
	for(i=0; i<4; i++) {
		for(j=0; j<4; j++) {
			// l_s1_d_weight[0][i][j] += l_s1_d_preact[m][h][w] * l_c1_output[m][h*4+i][w*4+j];
			// l_c1_d_output[m][h*4+i][w*4+j] += l_s1_weight[0][i][j] * l_s1_d_preact[m][h][w];

			atomicAdd(&l_s1_d_weight[0][i][j], l_s1_d_preact[m][h][w] * l_c1_output[m][h*4+i][w*4+j]);
			atomicAdd(&l_c1_d_output[m][h*4+i][w*4+j], l_s1_weight[0][i][j] * l_s1_d_preact[m][h][w]);
		}
	}

	if(m==0 && h<4 && w <4)
		l_s1_weight[0][h][w] += dt * l_s1_d_weight[0][h][w];
}


__global__ void bp_c1(
	float l_c1_preact[6][24][24],
	float l_c1_d_preact[6][24][24],
	float l_c1_d_output[6][24][24],
	float l_c1_d_weight[6][5][5],
	float l_c1_weight[6][5][5],
	float l_input_output[28][28],
	float l_c1_bias[6]

){
	// int n = blockIdx.x;
	int m = blockIdx.y;  // 6
	int h = threadIdx.x;  // 24
	int w = threadIdx.y;  // 24
	// int y = threadIdx.z;


	float o = sigmoid(l_c1_preact[m][h][w]);
	l_c1_d_preact[m][h][w] = l_c1_d_output[m][h][w] * o * (1 - o);

	int i, j;
	for(i=0; i<5; i++){
		for(j=0; j<5; j++){
			l_c1_d_weight[m][i][j] += l_c1_d_preact[m][h][w] * l_input_output[h + i][w + j] / (24*24);
		}
	}

	l_c1_bias[m] += dt * l_c1_d_preact[m][h][w] / (6*24*24);

	if(m==6 && h<5 && w <5)
		l_c1_weight[m][h][w] += dt * l_c1_d_weight[m][h][w];
}