#include "hip/hip_runtime.h"
#include "layer.h"



// Layer constructor:
Layer::Layer(int in_width, int in_height, int in_size): width(in_width), height(in_height), bytes(in_size){

    float h_bias[in_height];
    float h_weight[height][width];

    output = NULL;
    preact = NULL;
    bias = NULL:
    weight = NULL;

    for (int i = 0; i < height; i++){
        h_bias[i] = 0.5f - float(rand()) / float(RAND_MAX);  // initial bias
        for (int j = 0; j < width; j++){
            h_weight[i][j] = 0.5f - float(rand()) / float(RAND_MAX);  // initial weight
        }
    }

    hipMalloc(&output, sizeof(float) * bytes);
	hipMalloc(&preact, sizeof(float) * bytes);

	hipMalloc(&bias, sizeof(float) * N);
	hipMalloc(&weight, sizeof(float) * M * N);

	hipMalloc(&d_output, sizeof(float) * bytes);
	hipMalloc(&d_preact, sizeof(float) * bytes);
	hipMalloc(&d_weight, sizeof(float) * M * N);

	hipMemcpy(bias, h_bias, sizeof(float) * N, hipMemcpyHostToDevice);
	hipMemcpy(weight, h_weight, sizeof(float) * M * N, hipMemcpyHostToDevice);
}

// de-constructor
Layer::~Layer(){

    // TODO: free cuda memory
    hipFree(output);
	hipFree(preact);

	hipFree(bias);

	hipFree(weight);

	hipFree(d_output);
	hipFree(d_preact);
	hipFree(d_weight);

}

void Layer:: output(float *data){
    hipMemcpy(output, data, sizeof(float)*bytes, hipMemcpyHostToDevice);
}

void Layer:: clear(){

}
void Layer:: bp_clear(){

}


__device__ float sigmoid(float v){
    return 1/(1 + exp(-v));
}



