#include "hip/hip_runtime.h"
#include "layer.h"

#define MASK_WIDTH = 3;

// Layer constructor:
Layer::Layer(int in_width, int in_height, int in_size): M(in_width), N(in_height), bytes(in_size){

    float h_bias[N];
    float h_weight[N][M];


    output = NULL;
    preact = NULL;
    bias = NULL:
    weight = NULL;

    for (int i = 0; i < N; i++){
        h_bias[i] = 0.5f - float(rand()) / float(RAND_MAX);  // initial bias
        for (int j = 0; j < M; j++){
            h_weight[i][j] = 0.5f - float(rand()) / float(RAND_MAX);  // initial weight
        }
    }

    hipMalloc(&output, sizeof(float) * bytes);
	hipMalloc(&preact, sizeof(float) * bytes);

	hipMalloc(&bias, sizeof(float) * N);
	hipMalloc(&weight, sizeof(float) * M * N);

	hipMalloc(&d_output, sizeof(float) * bytes);
	hipMalloc(&d_preact, sizeof(float) * bytes);
	hipMalloc(&d_weight, sizeof(float) * M * N);

	hipMemcpy(bias, h_bias, sizeof(float) * N, hipMemcpyHostToDevice);
	hipMemcpy(weight, h_weight, sizeof(float) * M * N, hipMemcpyHostToDevice);
}

// de-constructor
Layer::~Layer(){

    // TODO: free cuda memory
    hipFree(output);
	hipFree(preact);

	hipFree(bias);

	hipFree(weight);

	hipFree(d_output);
	hipFree(d_preact);
	hipFree(d_weight);

}

void Layer:: setInput(float *data){
    hipMemcpy(output, data, sizeof(float)*bytes, hipMemcpyHostToDevice);
}

// void Layer:: clear(){

// }
// void Layer:: bp_clear(){

// }


__device__ float sigmoid(float v){
    return 1/(1 + exp(-v));
}

__global__ void apply_sigmoid(float *input, float *output, const int N){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int size = blockDim.x * gridDim.x;
    // TODO:

    for (int idx = N * pos / size; idx < N * (pos+1) / size; ++idx) {
		output[idx] = sigmoid(input[idx]);
	}


}

// convLayer 1 the weight is 6*3*3  output is 6*24*24
__global__ void ConvLayerForward_Kernel(int C, int W_grid, int K, float input[28][28], float output[6][24][24], float weight[6][5][5]){

    
    // int m, h, w, c, q, p;
    // float Pvalue = 0;
    // int n_start_point = i - (MASK_WIDTH/2);

    // for (int m = 0; m < 6; m++){          // for each output feature map
    //     for (int h = 0; h < 27; h++ ){
    //         for(int w = 0; w < 27; w++){
    //             output[m][h][w] = 0;
    //             for (int c = 0; c < 1; c++){ // for each channel
    //                 for (int p = 0; p < MASK_WIDTH; p++){
    //                     for (int q = 0; q < MASK_WIDTH; q++){
    //                         output[m][h][w] += input[h + p][w + q] * weight[m][p][q];
    //                     }
    //                 }
    //             }
    //         }

    //     }

    // }

    int m = blockIdx.x;
    int h =  blockIdx.y / W_grid  + threadIdx.y;
    int w = blockIdx.y % W_grid + threadIdx.x;
    float acc = 0.;
    //for (int c = 0;  c < C; c++) {		// sum over all input channels, in this case, the channel is 1
       for (int p = 0; p < K; p++)		// loop over KxK  filter
          for (int q = 0; q < K; q++)  
             acc += X[h+p][w+q] * weight[m][p][q];
    //}
    output[m][h][w] = acc;

}


