#include <windows.h>
#include <iostream>

#include <cstdlib>
#include <vector>
#include <memory>
#include <hipblas.h>
#include <hip/hip_runtime.h>

#include "layer.h"
#include "maxpooling.h"


// includes, system
#include <string>

// includes, kernels
#include <cnn_kernel.cu>


// poolingLayer_forward_GPU_naive<<<numBlocks,threadsPerBlock>>>(input_pointer, Inputimage_height,
//     Inputimage_width, Output_pointer, Outputimage_channel, pool_size);

