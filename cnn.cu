#include "hip/hip_runtime.h"
#define USE_MNIST_LOADER
#define MNIST_DOUBLE


// includes, system
#include <string>

#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <math.h>


#include "hip/hip_runtime.h"
#include ""

// //opencv for testing
// #include <opencv2/core/core.hpp>
// #include <opencv2/highgui/highgui.hpp>
// using namespace cv;

#include "mnist.h"
#include "layer.h"
#include "layer.cu"
//#include "maxpooling.h"
//#include "util.h"


//define the kernel size
#define TILE_WIDTH 16  //for small example


// set Layer
static Layer l_input = Layer(0, 0, 28*28);
static Layer l_c1 = Layer(5*5, 6, 24*24*6);
static Layer l_s1 = Layer(4*4, 1, 6*6*6);
static Layer l_f = Layer(6*6*6, 10, 10);

static mnist_data *train_set, *test_set;
static unsigned int train_cnt, test_cnt;


static inline void loadData(){
    mnist_load("MNIST_data/train-images.idx3-ubyte", "MNIST_data/train-labels.idx1-ubyte",
		&train_set, &train_cnt);
	mnist_load("MNIST_data/t10k-images.idx3-ubyte", "MNIST_data/t10k-labels.idx1-ubyte",
		&test_set, &test_cnt);
}



// void learn(){
   
// }

// void test(){
//     return 0;
// }


void forward(const double data[28][28]){

    printf("run forward\n");

    
    float input[28][28];

    for (int i = 0; i<28; i++){
        for (int j = 0; j<28; j++){
            input[i][j] = data[i][j];
            printf("%.2f ",data[i][j]);
        }
        printf("\n");
    }

    printf("**************************************\n");


    //example for convLayer 1:

    l_input.setInput((float *)input);

    //printf("input image: %f\n", &l_input.output[0][0]);
    

    int W_grid, H_grid;
    int W_out = 24, H_out = 24;
    int M = 6;  // The first (x) dimension in the grid maps to the M output feature maps
    W_grid = ceilf(W_out/TILE_WIDTH); 	// number of horizontal tiles per output map
    if (W_grid = 0) W_grid = 1;
    H_grid = H_out/TILE_WIDTH; 	// number of vertical tiles per output map
    int Y = H_grid * W_grid; //The second (y) dimension in the grid maps to the tiles in the output feature maps
    int C = 1, K = 5;
    dim3 blockDim(TILE_WIDTH, TILE_WIDTH, 1);
    // dim3 gridDim(M, Y, 1);
    // ConvLayerForward_Kernel<<< gridDim, blockDim>>>(int C = 1, W_grid, int K = 5, (float (*)[28])l_input.output,  (float (*)[24][24])l_c1.preact,(float (*)[5][5])l_c1.weight);
    // ConvLayerForward_Kernel_1<<< gridDim, blockDim>>>((float (*)[28])l_input.output,  (float (*)[24][24])l_c1.preact,(float (*)[5][5])l_c1.weight);


    int bz = ceil((float)28/TILE_WIDTH)*ceil((float)28/TILE_WIDTH);
    dim3 gridDim(1, 6, bz);

    ConvLayerForward_Kernel_1<<<gridDim,blockDim>>>((float (*)[28])l_input.output, (float (*)[24][24])l_c1.preact, (float (*)[5][5])l_c1.weight, 6, 28, 24, 24, 5, 6);
    //fp_preact_c1<<<64, 64>>>((float (*)[28])l_input.output, (float (*)[24][24])l_c1.preact, (float (*)[5][5])l_c1.weight);


    float *result = (float *)malloc(sizeof(float) * 24*24*6);

    hipMemcpy(result,
		l_c1.preact,
		24*24*6 * sizeof(float),
		hipMemcpyDeviceToHost);
    //apply_sigmoid <<<64,64>>>(l_c1.preact, l_c1.output, l_c1.size);

    for (int i = 0; i < 6; i++){
        for (int j = 0; j <24; j++){
            for (int z = 0; z < 24; z++){
                printf("%.2f",*(result + i+j+z));
            }
            printf("\n");
        }

        printf("-----------------------------------\n");
    }

    

    //write_ppm("test.ppm", 24, 24, 255, l_c1.preact[0]);

    //for sigmoid layer:
    // apply_sigmoid<<<64,64>>> (l_c1.preact, l_c1.output, l_c1.bytes);

    // MaxPool2dForward_Kernel_1<<<>>>((float(*)[24][24])l_c1.output, (float(*)[6][6]l_s1.preact, (float(*)[4][4]l_s1.weight)));
    // apply_sigmoid<<<64, 64>>>(l_s1.preact, l_s1.output, l_s1.bytes);


    // //for fully connected layer
    // FullyConLayerForward<<<64, 64>>>((float (*)[6][6])l_s1.output, l_f.preact, (float (*)[6][6][6])l_f.weight);
	// FullyConLayerForward_bias<<<64, 64>>>(l_f.preact, l_f.bias);
	// apply_sigmoid<<<64, 64>>>(l_f.preact, l_f.output, l_f.O);

                                               

}
// cnn -- iteration, lr
// int main(int argc, char** argv) {
//     // loaddata();
// 	learn();
// 	test();

// 	return 0;
// }




int main(){
    loadData();

    printf("test 2\n");
    forward(train_set[0].data);

    return 0;
}