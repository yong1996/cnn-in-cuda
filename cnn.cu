#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#define USE_MNIST_LOADER
#define MNIST_DOUBLE


// includes, system
#include <string>

#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <math.h>



// //opencv for testing
// #include <opencv2/core/core.hpp>
// #include <opencv2/highgui/highgui.hpp>
// using namespace cv;

#include "mnist.h"
#include "layer.h"
#include "layer.cu"
//#include "maxpooling.h"
//#include "util.h"


//define the kernel size
#define TILE_WIDTH 16  //for small example


// set Layer
static Layer l_input = Layer(0, 0, 28*28);
static Layer l_c1 = Layer(5*5, 6, 24*24*6);
static Layer l_s1 = Layer(4*4, 1, 6*6*6);
static Layer l_f = Layer(6*6*6, 10, 10);

static mnist_data *train_set, *test_set;
static unsigned int train_cnt, test_cnt;


static inline void loadData(){
    mnist_load("MNIST_data/train-images.idx3-ubyte", "MNIST_data/train-labels.idx1-ubyte",
		&train_set, &train_cnt);
	mnist_load("MNIST_data/t10k-images.idx3-ubyte", "MNIST_data/t10k-labels.idx1-ubyte",
		&test_set, &test_cnt);
}

void forward(const double data[28][28]){

    printf("run forward\n");

    
    float input[28][28];

    for (int i = 0; i<28; i++){
        for (int j = 0; j<28; j++){
            input[i][j] = data[i][j];
            printf("%.2f ",data[i][j]);
            // printf("%d ",(int)ceil(data[i][j]));
        }
        printf("\n");
    }

    printf("**************************************\n");


    //example for convLayer 1:

    l_input.setInput((float *)input);

    //printf("input image: %f\n", &l_input.output[0][0]);
    

    int W_grid, H_grid;
    int W_out = 24, H_out = 24;
    //int M = 6;  // The first (x) dimension in the grid maps to the M output feature maps
    W_grid = ceilf(W_out/TILE_WIDTH); 	// number of horizontal tiles per output map
    if (W_grid == 0) W_grid = 1;
    H_grid = H_out/TILE_WIDTH; 	// number of vertical tiles per output map
    //int Y = H_grid * W_grid; //The second (y) dimension in the grid maps to the tiles in the output feature maps
    //int C = 1, K = 5;
    int bz = ceil((float)28/TILE_WIDTH)*ceil((float)28/TILE_WIDTH);
    dim3 gridDim(1, 6, bz);
    dim3 blockDim(TILE_WIDTH, TILE_WIDTH, 1);

    ConvLayerForward_Kernel_1<<<gridDim,blockDim>>>((float (*)[28])l_input.output, (float (*)[24][24])l_c1.preact, (float (*)[5][5])l_c1.weight, 1, 28, 28, 24, 5, 6);

   


    // float *result = (float *)malloc(sizeof(float) * 24*24*6);

    // hipMemcpy(result,
	// 	l_c1.preact,
	// 	24*24*6 * sizeof(float),
	// 	hipMemcpyDeviceToHost);
    

    // for (int i = 0; i < 6; i++){
    //     for (int j = 0; j <24; j++){
    //         for (int z = 0; z < 24; z++){
    //             printf("%.2f",*(result + i+j+z));
    //         }
    //         printf("\n");
    //     }

    //     printf("-----------------------------------\n");
    // }

    apply_sigmoid <<<64,64>>>(l_c1.preact, l_c1.output, l_c1.bytes);


    // for pooling layer example:
    bz = ceil((float)6/TILE_WIDTH)*ceil((float)6/TILE_WIDTH);
    dim3 gridDimPool(1, 6, bz);
    dim3 blockDimPool(TILE_WIDTH, TILE_WIDTH, 1);
    MaxPool2dForward_Kernel_1<<<gridDimPool,blockDimPool>>>((float (*)[24][24])l_c1.output, (float (*)[6][6])l_s1.preact, 24, 24, 6, 4);
    apply_sigmoid <<<64,64>>>(l_s1.preact, l_s1.output, l_s1.bytes);




    // for fully connected layer
    bz = ceil((float)10/TILE_WIDTH);
    dim3 gridDimfc(1, 6, bz);
    dim3 blockDimfc(TILE_WIDTH, TILE_WIDTH, 1);

    FullyConLayerForward_kernel<<<gridDimfc,blockDimfc>>>((float (*)[6][6])l_s1.output, (float (*)[6][6][6])l_f.weight, l_f.preact, l_f.bias, 1, 6, 10, 1, 10);
    // FullyConLayerForward_kernel<<<gridDimfc,blockDimfc>>>(X_pointer, (float (*)[24][24])l_c1.preact, (float (*)[5][5])l_c1.weight, (float *)l_c1.bias, 28, 28, 24, 6, 4);
    apply_sigmoid <<<64,64>>>(l_f.preact, l_f.output, l_f.bytes);
    // softmax<<<10,1>>>(l_f.d_preact, l_f.output, train_set[i].label, 10);


    float *result = (float *)malloc(sizeof(float) * 10);

    hipMemcpy(result, l_f.preact, 10 * sizeof(float), hipMemcpyDeviceToHost);
    

    printf("ConvLayerForward_Kernel: \n");
    for (int i = 0; i < 10; i++){
        printf("%.2f ",*(result + i));
    }
    printf("\n-----------------------------------\n");



}

void backward(){
    int bz = ceil((float)10/TILE_WIDTH);
    dim3 gridDimfc(1, 6, bz);
    dim3 blockDimfc(10, 10, 1);
    FullyConLayerBackward_kernel<<<gridDimfc,blockDimfc>>>((float (*)[6][6])l_s1.output, (float (*)[6][6][6])l_f.weight, l_f.preact, l_f.bias, 1, 6, 10);
    
    // bz = ceil((float)6/TILE_WIDTH)*ceil((float)6/TILE_WIDTH);
    // dim3 gridDimPool(1, 6, bz);
    // dim3 blockDimPool(TILE_WIDTH, TILE_WIDTH, 1);
    // MaxPool2dBackward_Kernel<<<gridDimPool,blockDimPool>>>((float (*)[24][24])l_c1.output, (float (*)[6][6])l_s1.preact, 24, 24, 6, 4);
    // MaxPool2dBackward_kernel<<<64, 64>>>((float (*)[6][6][6])l_f.d_weight, l_f.d_preact, (float (*)[6][6])l_s1.output);
    
    bz = ceil((float)28/TILE_WIDTH)*ceil((float)28/TILE_WIDTH);
    dim3 gridDim(1, 6, bz);
    dim3 blockDim(TILE_WIDTH, TILE_WIDTH, 1);
    ConvLayerBackward_Kernel<<<gridDim,blockDim>>>((float (*)[28])l_input.output, (float (*)[24][24])l_c1.preact, (float (*)[5][5])l_c1.weight, l_c1.bias, 1, 24, 24, 6, 5, 6);
    // ConvLayerwbBackward_Kernel<<<gridDim,blockDim>>>((float (*)[24][24])l_c1.d_output, (float (*)[24][24])l_c1.d_preact, (float (*)[4][4])l_c1.d_weight, l_c1.bias, 1, 28, 28, 24, 5, 6);

    // ConvLayerBackward_kernel<<<64, 64>>>((float (*)[6][6][6])l_f.d_weight, l_f.d_preact, (float (*)[6][6])l_s1.output);
}


int main(){
    loadData();

    printf("test 666\n");
    forward(train_set[0].data);

    backward();
    
    printf("finish\n");

    return 0;
}